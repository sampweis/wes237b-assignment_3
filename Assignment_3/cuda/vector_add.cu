
#include <hip/hip_runtime.h>
#include "stdio.h"

#define N 10000000


__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = threadIdx.x; i < n; i += blockDim.x){
        out[i] = a[i] + b[i];
    }
}

__global__ void vector_add_block(float *out, float *a, float *b, int n) {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    out[idx] = a[idx] + b[idx];
}


int main(){
    float *a, *b, *out;
    
    float *d_a, *d_b, *d_out;

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }
    
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Main function
    //vector_add<<<1,256>>>(d_out, d_a, d_b, N);
    
    int block_size = 256;
    int grid_size = ((N + block_size) / block_size);
    vector_add_block<<<grid_size,block_size>>>(d_out, d_a, d_b, N);
    
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    printf("val: %f, val[255]: %f, val[256]: %f\n", *out, *(out+255), *(out+256));
}
